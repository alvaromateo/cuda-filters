/*

**************************************************************************

cuda-filters == command line program to apply various filters to images
Copyright (C) 2016  Alvaro Mateo (alvaromateo9@gmail.com)
					Biel Pieras (bpierasmorell@gmail.com)

**************************************************************************

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.
  
***************************************************************************

*/

// Includes
#include <math.h>

extern "C" {
	#include "readCommandLine.h"
}

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

float avg3[9] = {1./9, 1./9, 1./9, 1./9, 1./9, 1./9, 1./9, 1./9, 1./9};
float avg5[25] = {1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25};
float sharpenWeak[9] = {0,-1,0,-1,5,-1,0,-1,0};
float sharpenStrong[9] = {-1,-1,-1,-1,9,-1,-1,-1,-1};
float gaussian3[9] = {1./16, 2./16, 1./16, 2./16, 4./16, 2./16, 1./16, 2./16, 1./16};
float gaussian5[25] = {1./256, 4./256, 6./256, 4./256, 1./256, 4./256, 16./256, 24./256, 16./256, 4./256, 6./256, 24./256, 36./256, 24./256, 6./256, 4./256, 16./256, 24./256, 16./256, 4./256, 1./256, 4./256, 6./256, 4./256, 1./256};
float edgeDetection[9] = {0,1,0,1,-4,1,0,1,0}; //Normalize result by adding 128 to all elements
float embossing[9] = {-2,-1,0,-1,1,1,0,1,2};

// Filter array
float *arrayFilter[] = {&avg3[0], &avg5[0], &sharpenWeak[0], &sharpenStrong[0], &gaussian3[0], &gaussian5[0], &edgeDetection[0], &embossing[0]};

// Methods
uchar getFiltersize(uchar filterType) {
	uchar filterSize = 3;
	switch (filterType) {
		case 1:
		case 5:
			filterSize = 5;
			break;
	}
	return filterSize;
}


int main(int argc, char **argv) {
	// Initialize options
	uchar filterType, threads, pinned;
    char *imageName = getOptions(argc, argv, &filterType, &threads, &pinned);

    // bitDepth has the number of channels: 1 for grayscale and 3 for RGB
	int width, height, bitDepth;
	uchar *image = stbi_load(imageName, &width, &height, &bitDepth, 0);

    // Check for invalid input
    if ( image == NULL ) {
        printf("Could not open or find the image\n");
        return -1;
    }

    uint color = !(bitDepth % 2) ? (bitDepth - 1) : bitDepth; // with this we ignore the alpha channel

	/*
	 * Start kernel part!
	 */

	// Pointers to variables in the host
    uchar **channels = (uchar **) malloc(color * sizeof(uchar *));
    uchar **output = (uchar **) malloc(color * sizeof(uchar *));
    // Pointers to variables in the device
    uchar **channelsDevice = (uchar **) malloc(color * sizeof(uchar *));
    uchar **outputDevice = (uchar **) malloc(color * sizeof(uchar *));
    
	//Separate the channels
	uint i, j, x;
	uint len = width * height;
	uint numBytesImage = len * sizeof(uchar);

	for (x = 0; x < color; ++x) {
		if (pinned) {
			hipHostMalloc((uchar **) &channels[x], numBytesImage);
			hipHostMalloc((uchar **) &output[x], numBytesImage);
		} else {
			channels[x] = (uchar *) malloc(len * sizeof(uchar));
			output[x] = (uchar *) malloc(len * sizeof(uchar));
		}
	}
	
	// Initialize matrixs
	for (i = 0, j = 0; i < bitDepth*len; i += bitDepth, ++j){
		for (x = 0; x < color; ++x) { // we leave the alpha channel unchanged
			(channels[x])[j] = image[i + x];
			(output[x])[j] = image[i + x];
		}
	}

	// Get filter
	float *filter, *filterDevice;
	uint filterX, filterY, filterSize, numBytesFilter;

	// Initialize filterSize
    filterSize = getFiltersize(filterType);
    numBytesFilter = filterSize * filterSize * sizeof(float);

	if (pinned) {
		hipHostMalloc((float **) &filter, numBytesFilter);
	} else {
		filter = arrayFilter[filterType];
	}

    // Variables to calculate time spent in each job
	float TiempoTotal, TiempoKernel;
	hipEvent_t E0, E1, E2, E3;

	// Number of blocks in each dimension 
	uint nBlocksX = (width + threads - 1) / threads; 
	uint nBlocksY = (height + threads - 1) / threads;

	dim3 dimGrid(nBlocksX, nBlocksY, 1);
	dim3 dimBlock(threads, threads, 1);

	hipEventCreate(&E0);
	hipEventCreate(&E1);
	hipEventCreate(&E2);
	hipEventCreate(&E3);

	hipEventRecord(E0, 0);
	hipEventSynchronize(E0);

	// Get memory in device
	// Filter
	hipMalloc((float**) &filterDevice, numBytesFilter); 
	// Image
	hipMalloc((uchar**)&iRed, numBytesImage); 
	hipMalloc((uchar**)&iGreen, numBytesImage); 
	hipMalloc((uchar**)&iBlue, numBytesImage); 
	// Output image
	hipMalloc((uchar**)&iModRed, numBytesImage); 
	hipMalloc((uchar**)&iModGreen, numBytesImage); 
	hipMalloc((uchar**)&iModBlue, numBytesImage); 

	// Copy data from host to device 
	hipMemcpy(f, f_H, numBytesFilter, hipMemcpyHostToDevice);
	hipMemcpy(iRed, iRed_H, numBytesImage, hipMemcpyHostToDevice);
	hipMemcpy(iGreen, iRed_H, numBytesImage, hipMemcpyHostToDevice);
	hipMemcpy(iBlue, iRed_H, numBytesImage, hipMemcpyHostToDevice);

	hipEventRecord(E1, 0);
	hipEventSynchronize(E1);

	// Execute the kernel
	kernel<<<dimGrid, dimBlock>>>(filter.getWidth(), filter.getWidth() / 2, image.getWidth(), image.getHeight(), f, iRed, iModRed);
	kernel<<<dimGrid, dimBlock>>>(filter.getWidth(), filter.getWidth() / 2, image.getWidth(), image.getHeight(), f, iGreen, iModGreen);
	kernel<<<dimGrid, dimBlock>>>(filter.getWidth(), filter.getWidth() / 2, image.getWidth(), image.getHeight(), f, iBlue, iModBlue);

	//recordEvent(E2);
	hipEventRecord(E2, 0);
	hipEventSynchronize(E2);

	// Get the result to the host 
	hipMemcpy(iModRed_H, iModRed, numBytesImage, hipMemcpyDeviceToHost); 
	hipMemcpy(iModGreen_H, iModGreen, numBytesImage, hipMemcpyDeviceToHost);
	hipMemcpy(iModBlue_H, iModBlue, numBytesImage, hipMemcpyDeviceToHost);

	// Copy the result to image
	image[0].setMatrix(iModRed_H);
	image[1].setMatrix(iModGreen_H);
	image[2].setMatrix(iModBlue_H);

	// Free memory of the device 
	hipFree(f);
	hipFree(iRed);
	hipFree(iGreen);
	hipFree(iBlue);
	hipFree(iModRed);
	hipFree(iModGreen);
	hipFree(iModBlue);

	//recordEvent(E3);
	hipEventRecord(E3, 0);
	hipEventSynchronize(E3);

	hipEventElapsedTime(&TiempoTotal,  E0, E3);
	hipEventElapsedTime(&TiempoKernel, E1, E2);

	// Print results. TODO

	hipEventDestroy(E0);
	hipEventDestroy(E1);
	hipEventDestroy(E2);
	hipEventDestroy(E3);

	if (pinned) {
		hipHostFree(f_H);
		hipHostFree(iRed_H);
		hipHostFree(iGreen_H);
		hipHostFree(iBlue_H);
		hipHostFree(iModRed_H);
		hipHostFree(iModGreen_H);
		hipHostFree(iModBlue_H);
	} else {
		free(f_H);
		free(iRed_H);
		free(iGreen_H);
		free(iBlue_H);
		free(iModRed_H);
		free(iModGreen_H);
		free(iModBlue_H);
	}

	/*
	 * End kernel part!
	 */

    // Write the image to disk appending "_filter" to its name
	char newImageName[NAME_SIZE] = "\0";
	strncpy(newImageName, imageName, strlen(imageName) - 4);
	strncat(newImageName, "_filter.png", NAME_SIZE - strlen(newImageName) - 1);
	stbi_write_png(newImageName, width, height, bitDepth, image, width * bitDepth);

    return 0;
}
