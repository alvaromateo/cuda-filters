#include "hip/hip_runtime.h"
/*

**************************************************************************

cuda-filters == command line program to apply various filters to images
Copyright (C) 2016  Alvaro Mateo (alvaromateo9@gmail.com)
					Biel Pieras (bpierasmorell@gmail.com)

**************************************************************************

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.
  
***************************************************************************

*/

// Includes
#include <math.h>

extern "C" {
	#include "readCommandLine.h"
}

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

float avg3[9] = {1./9, 1./9, 1./9, 1./9, 1./9, 1./9, 1./9, 1./9, 1./9};
float avg5[25] = {1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25, 1./25};
float sharpenWeak[9] = {0,-1,0,-1,5,-1,0,-1,0};
float sharpenStrong[9] = {-1,-1,-1,-1,9,-1,-1,-1,-1};
float gaussian3[9] = {1./16, 2./16, 1./16, 2./16, 4./16, 2./16, 1./16, 2./16, 1./16};
float gaussian5[25] = {1./256, 4./256, 6./256, 4./256, 1./256, 4./256, 16./256, 24./256, 16./256, 4./256, 6./256, 24./256, 36./256, 24./256, 6./256, 4./256, 16./256, 24./256, 16./256, 4./256, 1./256, 4./256, 6./256, 4./256, 1./256};
float edgeDetection[9] = {0,1,0,1,-4,1,0,1,0}; //Normalize result by adding 128 to all elements
float embossing[9] = {-2,-1,0,-1,1,1,0,1,2};

// Filter array
float *arrayFilter[] = {&avg3[0], &avg5[0], &sharpenWeak[0], &sharpenStrong[0], &gaussian3[0], &gaussian5[0], &edgeDetection[0], &embossing[0]};

// Methods
uchar getFiltersize(uchar filterType) {
	uchar filterSize = 3;
	switch (filterType) {
		case 1:
		case 5:
			filterSize = 5;
			break;
	}
	return filterSize;
}

void initFilter(float *filter, uint filterSize, uchar filterType) {
	for (uint a = 0; a < filterSize; ++a) {
		filter[a] = (arrayFilter[filterType])[a];
	}
}

void CheckCudaError(char sms[], int line) {
	hipError_t error;
	error = hipGetLastError();
	if (error) {
		printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
		exit(EXIT_FAILURE);
	}
}

__global__ void kernel(int width, int height, int filterSize, float *filt, uchar *img, uchar *out) {
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	uint j = blockIdx.y * blockDim.y + threadIdx.y;
	uint padding = filterSize / 2;
	unsigned long int index = i * width + j;

	if ((i > padding) && (j > padding) && (i < width - padding) && (j < height - padding)) {
		float tmp = 0.0;
		for (uint filterX = 0; filterX < filterSize; ++filterX) {
			for (uint filterY = 0; filterY < filterSize; ++filterY) {
				uint imageX = (i - padding + filterX);
				uint imageY = (j - padding + filterY);
				tmp += ((float) img[imageX * width + imageY] * (float) filt[filterX * filterSize + filterY]);
			}
		}
		out[index] = (uchar) (tmp < 0) ? 0 : ((tmp > 255) ? 255 : tmp);
	}
}


int main(int argc, char **argv) {
	// Initialize options
	uchar filterType, threads, pinned;
    char *imageName = getOptions(argc, argv, &filterType, &threads, &pinned);

    // bitDepth has the number of channels: 1 for grayscale and 3 for RGB
	int width, height, bitDepth;
	uchar *image = stbi_load(imageName, &width, &height, &bitDepth, 0);

    // Check for invalid input
    if ( image == NULL ) {
        printf("Could not open or find the image\n");
        return -1;
    }

    uint color = !(bitDepth % 2) ? (bitDepth - 1) : bitDepth; // with this we ignore the alpha channel

	/*
	 * Start kernel part!
	 */

	// Pointers to variables in the host
    uchar **channels = (uchar **) malloc(color * sizeof(uchar *));
    // Pointers to variables in the device
    uchar **channelsDevice = (uchar **) malloc(color * sizeof(uchar *));
    uchar **outputDevice = (uchar **) malloc(color * sizeof(uchar *));
    
	//Separate the channels
	uint i, j, x;
	uint len = width * height;
	uint numBytesImage = len * sizeof(uchar);

	for (x = 0; x < color; ++x) {
		if (pinned) {
			hipHostMalloc((uchar **) &channels[x], numBytesImage);
			CheckCudaError((char *) "Obtener Memoria en el host", __LINE__);
		} else {
			channels[x] = (uchar *) malloc(len * sizeof(uchar));
		}
	}
	
	// Initialize matrixs
	for (i = 0, j = 0; i < bitDepth*len; i += bitDepth, ++j){
		for (x = 0; x < color; ++x) { // we leave the alpha channel unchanged
			(channels[x])[j] = image[i + x];
		}
	}

	// Get filter
	float *filter, *filterDevice;
	uint filterSize, numBytesFilter;

	// Initialize filterSize
    filterSize = getFiltersize(filterType);
    numBytesFilter = filterSize * filterSize * sizeof(float);

	if (pinned) {
		hipHostMalloc((float **) &filter, numBytesFilter);
		CheckCudaError((char *) "Obtener Memoria en el host", __LINE__);
	} else {
		filter = (float *) malloc(filterSize * sizeof(float));
	}
	initFilter(filter, filterSize, filterType);

    // Variables to calculate time spent in each job
	float TiempoTotal, TiempoKernel;
	hipEvent_t E0, E1, E2, E3;

	// Number of blocks in each dimension 
	uint nBlocksX = (width + threads - 1) / threads; 
	uint nBlocksY = (height + threads - 1) / threads;

	dim3 dimGrid(nBlocksX, nBlocksY, 1);
	dim3 dimBlock(threads, threads, 1);

	hipEventCreate(&E0);
	hipEventCreate(&E1);
	hipEventCreate(&E2);
	hipEventCreate(&E3);

	hipEventRecord(E0, 0);
	hipEventSynchronize(E0);

	// Get memory in device and send data
	// Filter
	hipMalloc((float**) &filterDevice, numBytesFilter); 
	CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);
	hipMemcpy(filterDevice, filter, numBytesFilter, hipMemcpyHostToDevice);
	CheckCudaError((char *) "Copiar Memoria en el device", __LINE__);
	// Image
	for (x = 0; x < color; ++x) {
		hipMalloc((uchar **) &channelsDevice[x], numBytesImage);
		CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);
		hipMalloc((uchar **) &outputDevice[x], numBytesImage);
		CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);
		hipMemcpy(channelsDevice[x], channels[x], numBytesImage, hipMemcpyHostToDevice);
		CheckCudaError((char *) "Copiar Memoria en el device", __LINE__);
	}

	hipEventRecord(E1, 0);
	hipEventSynchronize(E1);

	// Execute the kernel
	for (x = 0; x < color; ++x) {
		kernel<<<dimGrid, dimBlock>>>(width, height, filterSize, filterDevice, channelsDevice[x], outputDevice[x]);
	}
	printf("After the kernel\n");
	
	//recordEvent(E2);
	hipEventRecord(E2, 0);
	hipEventSynchronize(E2);

	// Get the result to the host and free memory
	hipFree(filterDevice);
	CheckCudaError((char *) "Liberar Memoria en el device", __LINE__);
	for (x = 0; x < color; ++x) {
		hipMemcpy(channels[x], outputDevice[x], numBytesImage, hipMemcpyDeviceToHost);
		CheckCudaError((char *) "Copiar Memoria en el host", __LINE__);
		hipFree(channelsDevice[x]);
		CheckCudaError((char *) "Liberar Memoria en el device", __LINE__);
		hipFree(outputDevice[x]);
		CheckCudaError((char *) "Liberar Memoria en el device", __LINE__);
	}

	//recordEvent(E3);
	hipEventRecord(E3, 0);
	hipEventSynchronize(E3);

	hipEventElapsedTime(&TiempoTotal,  E0, E3);
	hipEventElapsedTime(&TiempoKernel, E1, E2);

	// Print results. TODO

	hipEventDestroy(E0);
	hipEventDestroy(E1);
	hipEventDestroy(E2);
	hipEventDestroy(E3);

	// Rejoin the channels to save the image
    for (i = 0, j = 0; i < bitDepth*len; i += bitDepth, ++j){
		for (x = 0; x < color; ++x) { // we leave the alpha channel unchanged
			image[i + x] = (channels[x])[j];
		}
	}

	// Free memory of the host
	if (pinned) {
		hipHostFree(filter);
	} else {
		free(filter);
	}
	for (x = 0; x < color; ++x) {
		if (pinned) {
			hipHostFree(channels[x]);
		} else {
			free(channels[x]);
		}
	}
	free(channels);
	free(channelsDevice);
	free(outputDevice);

	/*
	 * End kernel part!
	 */


	// Write the image to disk appending "_filter" to its name
	char newImageName[NAME_SIZE] = "\0";
	strncpy(newImageName, imageName, strlen(imageName) - 4);
	strncat(newImageName, "_filter.png", NAME_SIZE - strlen(newImageName) - 1);
	stbi_write_png(newImageName, width, height, bitDepth, image, width * bitDepth);

    return 0;
}
